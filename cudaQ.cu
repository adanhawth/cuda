/*
 *	Returns the abbreviated property's list for all CUDA devices.
 */

#include <stdio.h>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("\nnvcc (CUDA toolkit) version: %d.%d.%d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);

    int driverVersion;
    if (hipDriverGetVersion(&driverVersion) == hipSuccess)
        printf("CUDA driver version: %d.%d\n", driverVersion/1000, (driverVersion%100)/10);
    else
        printf("CUDA driver version: NA\n");

    int runtimeVersion;
    if (hipRuntimeGetVersion(&runtimeVersion) == hipSuccess)
        printf("CUDA runtime version: %d.%d\n", runtimeVersion/1000, (runtimeVersion%100)/10);
    else
        printf("CUDA runtime version: NA\n");


    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        printf("\nDevice Number: %d\n", i);
        printf("  Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Device name: %s\n", deviceProp.name);
        printf("  Memory Clock Rate (KHz): %d\n", deviceProp.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", deviceProp.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0 * deviceProp.memoryClockRate * (deviceProp.memoryBusWidth / 8.0) / 1.0e6);
        printf("  Total amount of shared memory per block: %lu\n", deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    }
    printf("\n");
}